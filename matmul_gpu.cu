#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matmul_cpu.c"

__global__ void matmul_kernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void random_matrix(float* M, int N) {
    for (int i = 0; i < N * N; i++) {
        M[i] = (float)(rand() % 100) / 10.0f;
    }
}

int main() {
    int N = 512; // test size
    size_t size = N * N * sizeof(float);

    // allocate host memory (cpu)
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    random_matrix(h_A, N);
    random_matrix(h_B, N);

    // allocate device memory (gpu)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // copy to device (gpu)
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // kernel launch
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N+15)/16, (N+15)/16);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // back to cpu
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("GPU time (N=%d): %f ms\n", N, ms);

    // cpu check
    float* h_C_cpu = (float*)malloc(size);

    clock_t start_cpu = clock();
    matmul_cpu(h_A, h_B, h_C_cpu, N);
    clock_t end_cpu = clock();
    double cpu_time = 1000.0 * (end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("CPU time (N=%d): %f ms\n", N, cpu_time);
    
    // memory deallocation
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);





    return 0;
}
